#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include "const.h"


/**
 * @note This is function called from GPU and executed on GPU
 * @param p - first point
 * @param p2 - second point
 * @return the distance between two points
 */
__device__ double kernel_distance(const Point* p, const Point* p2)
{
    double x_square, y_square, res;
    x_square = pow(p->x - p2->x, 2);
    y_square = pow(p->y - p2->y, 2);
    res = sqrt(x_square + y_square);
    return  res;
}

/**
 * @brief This function calculating weather there are K points with distance greater than D
 * @note This function called from CPU and executed on GPU
 * @note function using atomic add to avoid adding at the same time
 * @param points_arr - an arr of points
 * @param N - arr size
 * @param K - number of points that supposed to be in a distance greater than D
 * @param D - distance to be greater than
 * @param point_index - point index to find if there are K different points with distance greater than D
 * @param points_range_count - pointer for returned value, weather there are or aren't K points
 */
__global__ void kernel_function(const Point* points_arr, int N, int K, double D,
                                int point_index, int* points_range_count)
{
    int i  = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && i != point_index && *points_range_count < K)
    {
        if (kernel_distance(&points_arr[point_index], &points_arr[i]) < D)
        {
            atomicAdd(points_range_count,1);
        }
    }
}

/**
 * @brief a malloc for cuda function that makes the test of failed allocation
 * @note case this function fails - it aborts the program
 * @param size - number of bytes to be malloced on GPU
 * @return a pointer for the malloced memory on the GPU
 */
void* my_cuda_malloc(size_t size)
{
    void* ptr = NULL;
    hipError_t err;

    err = hipMalloc((void **)&ptr, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return ptr;
}

/**
 * @brief a memory copy for cuda function that makes the test of failed allocation
 * @note case this function fails - it aborts the program
 * @param dst - a pointer for the destination
 * @param src - a pointer for the source
 * @param count - how many bytes to be copied
 * @param kind - weather it's from host to device or device to host
 */
void my_cuda_mem_cpy(void* dst, const void* src, size_t count, hipMemcpyKind kind)
{
    hipError_t err;
    err = hipMemcpy(dst, src, count, kind);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from %s - %s\n",
                (kind == hipMemcpyHostToDevice) ? "host to device" : "device to host",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * @brief a free memory cuda function that makes the test of failed allocation
 * @note case this function fails - it aborts the program
 * @param ptr - the pointer to be free on the GPU
 */
void my_cuda_free(void* ptr)
{
    hipError_t err;
    err = hipFree(ptr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * @brief this function validate weather there are 3 points that satisfied problem definition using cuda
 * @note This function using GPU implementation. Case something goes wrong - it aborts the program
 * @param points_arr - an array of points
 * @param N - the size of the array
 * @param K - number of points that supposed to be in a distance greater than D
 * @param D - distance to be greater than
 * @param local_results - sub array of desired points (K different points greater than distance D)
 * @param index_local_results - index of the local_results array
 * @return True if there are at least SATISFIED_IDS_SIZE that fit the problem definition. False otherwise.
 */
Boolean compute_CUDA(const Point *points_arr, int N, int K, double D,
                     PointProCent* local_results, int index_local_results)
{
    int i, points_range_count, points_satisfied_pro_count, threadsPerBlock, blocksPerGrid;
    int* d_points_range_count;
    size_t arr_size;
    Point *cuda_arr = NULL;
    hipError_t err;

    // Define GPU variables
    arr_size = N * sizeof(Point);
    cuda_arr = (Point *) my_cuda_malloc(arr_size);
    d_points_range_count = (int*) my_cuda_malloc(sizeof(int));
    my_cuda_mem_cpy(cuda_arr, points_arr, arr_size, hipMemcpyHostToDevice);

    // Define the number of threads to work simultaneously
    threadsPerBlock = MAX_THREADS_PER_BLOCK;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    points_satisfied_pro_count = 0;

    for(i = 0; i < N && points_satisfied_pro_count < SATISFIED_IDS_SIZE; ++i)
    {
        // Set memory to GPU
        points_range_count = 0;
        my_cuda_mem_cpy(d_points_range_count, &points_range_count,
                        sizeof(int), hipMemcpyHostToDevice);

        kernel_function<<<blocksPerGrid, threadsPerBlock>>>
                (cuda_arr, N, K, D, i,d_points_range_count);

        // Validate kernel function worked properly
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, ERROR_KERNEL_FORMAT, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Get results from GPU
        my_cuda_mem_cpy(&points_range_count, d_points_range_count,
                        sizeof(int), hipMemcpyDeviceToHost);

        if(points_range_count >= K)
        {
            local_results[index_local_results].points_index[points_satisfied_pro_count++] = points_arr[i].Nid;
        }

    }

    // Free allocated memory on GPU
    my_cuda_free(cuda_arr);
    my_cuda_free(d_points_range_count);

    return (points_satisfied_pro_count >= SATISFIED_IDS_SIZE) ? True : False;

}